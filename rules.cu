#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#ifdef _WIN32
#define DLL_EXPORT __declspec(dllexport)
#else
#define DLL_EXPORT __attribute__((visibility("default")))
#endif

#define MAX_LEN 32
#define THREADS 512
// nvcc --shared -o rules.dll rules.cu -Xcompiler "/MD" -link -cudart static
// nvcc -shared -o librules.dll rules.cu

// XXHash64 constants
#define PRIME64_1 0x9E3779B185EBCA87ULL
#define PRIME64_2 0xC2B2AE3D27D4EB4FULL
#define PRIME64_3 0x165667B19E3779F9ULL
#define PRIME64_4 0x85EBCA77C2B2AE63ULL
#define PRIME64_5 0x27D4EB2F165667C5ULL
#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", \
                hipGetErrorString(err), __FILE__, __LINE__); \
        exit(1); \
    } \
} while(0)

__device__ bool binarySearchGPU(const uint64_t* array, uint64_t length, uint64_t target) {
    int left = 0;
    int right = length - 1;
    while (left <= right) {
        int mid = left + (right - left) / 2;
        uint64_t midVal = array[mid];
        if (midVal == target) return true;
        if (midVal < target) left = mid + 1;
        else right = mid - 1;
    }
    return false;
}

// 64-bit rotation function
__device__ uint64_t rotl64(uint64_t x, int r) {
    return (x << r) | (x >> (64 - r));
}

// XXHash round operation
__device__ uint64_t round(uint64_t acc, uint64_t input) {
    acc += input * PRIME64_2;
    acc = rotl64(acc, 31);
    acc *= PRIME64_1;
    return acc;
}

// Merge round for combining accumulators
__device__ uint64_t mergeRound(uint64_t acc, uint64_t val) {
    val = round(0, val);
    acc ^= val;
    acc = acc * PRIME64_1 + PRIME64_4;
    return acc;
}

// Read 32-bit value from unaligned memory
__device__ uint32_t read32(const void* ptr) {
    const uint8_t* byte_ptr = (const uint8_t*)ptr;
    return (uint32_t)byte_ptr[0] | ((uint32_t)byte_ptr[1] << 8) |
           ((uint32_t)byte_ptr[2] << 16) | ((uint32_t)byte_ptr[3] << 24);
}

// Read 64-bit value from unaligned memory (little-endian)
__device__ uint64_t read64(const void* ptr) {
    const uint8_t* byte_ptr = (const uint8_t*)ptr;
    return (uint64_t)byte_ptr[0] | ((uint64_t)byte_ptr[1] << 8) |
           ((uint64_t)byte_ptr[2] << 16) | ((uint64_t)byte_ptr[3] << 24) |
           ((uint64_t)byte_ptr[4] << 32) | ((uint64_t)byte_ptr[5] << 40) |
           ((uint64_t)byte_ptr[6] << 48) | ((uint64_t)byte_ptr[7] << 56);
}

// XXHash64 computation for a single string
__device__ uint64_t xxhash64(const char* input, int len, uint64_t seed) {
    const char* p = input;
    const char* const end = p + len;
    uint64_t h64;

    if (len >= 32) {
        const char* const limit = end - 32;
        uint64_t v1 = seed + PRIME64_1 + PRIME64_2;
        uint64_t v2 = seed + PRIME64_2;
        uint64_t v3 = seed + 0;
        uint64_t v4 = seed - PRIME64_1;

        // Process 32-byte chunks
        do {
            v1 = round(v1, read64(p)); p += 8;
            v2 = round(v2, read64(p)); p += 8;
            v3 = round(v3, read64(p)); p += 8;
            v4 = round(v4, read64(p)); p += 8;
        } while (p <= limit);

        // Combine accumulators
        h64 = rotl64(v1, 1) + rotl64(v2, 7) + rotl64(v3, 12) + rotl64(v4, 18);
        h64 = mergeRound(h64, v1);
        h64 = mergeRound(h64, v2);
        h64 = mergeRound(h64, v3);
        h64 = mergeRound(h64, v4);
    } else {
        h64 = seed + PRIME64_5;
    }

    h64 += (uint64_t)len;

    // Process remaining bytes
    while (p + 8 <= end) {
        uint64_t k1 = read64(p);
        k1 *= PRIME64_2;
        k1 = rotl64(k1, 31);
        k1 *= PRIME64_1;
        h64 ^= k1;
        h64 = rotl64(h64, 27) * PRIME64_1 + PRIME64_4;
        p += 8;
    }

    // Process last 4 bytes
    if (p + 4 <= end) {
        h64 ^= (uint64_t)read32(p) * PRIME64_1;
        h64 = rotl64(h64, 23) * PRIME64_2 + PRIME64_3;
        p += 4;
    }

    // Process remaining bytes
    while (p < end) {
        h64 ^= (*p) * PRIME64_5;
        h64 = rotl64(h64, 11) * PRIME64_1;
        p++;
    }

    // Final avalanche
    h64 ^= h64 >> 33;
    h64 *= PRIME64_2;
    h64 ^= h64 >> 29;
    h64 *= PRIME64_3;
    h64 ^= h64 >> 32;

    return h64;
}

// CUDA kernel to compute hashes for all strings
extern "C" {
// 'l' Rule: converts all (ASCII) characters to lowercase.
__global__ void lowerCaseKernel(char *words, int *lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char *word = &words[idx * MAX_LEN];
        for (int i = 0; i < len; i++) {
            // Simple ASCII conversion: if character is A-Z, add 32.
            if (word[i] >= 'A' && word[i] <= 'Z') {
                word[i] = word[i] + 32;
            }
        }
    }
}

// 'u' Rule: converts all (ASCII) characters to uppercase.
__global__ void upperCaseKernel(char *words, int *lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char *word = &words[idx * MAX_LEN];
        for (int i = 0; i < len; i++) {
            // Simple ASCII conversion: if character is a-z, subtract 32.
            if (word[i] >= 'a' && word[i] <= 'z') {
                word[i] = word[i] - 32;
            }
        }
    }
}

// 'c' Rule: Capitalize first letter, lowercase rest
__global__ void capitalizeKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0) return;

        // Lowercase all characters
        for (int i = 0; i < len; ++i) {
            if (word[i] >= 'A' && word[i] <= 'Z') {
                word[i] += 32;
            }
        }
        // Uppercase first character
        if (word[0] >= 'a' && word[0] <= 'z') {
            word[0] -= 32;
        }
    }
}

// 'C' Rule: Lowercase first letter, uppercase rest
__global__ void invertCapitalizeKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0) return;

        // Uppercase all characters
        for (int i = 0; i < len; ++i) {
            if (word[i] >= 'a' && word[i] <= 'z') {
                word[i] -= 32;
            }
        }
        // Lowercase first character
        if (word[0] >= 'A' && word[0] <= 'Z') {
            word[0] += 32;
        }
    }
}

// 't' Rule: Toggle case of all characters
__global__ void toggleCaseKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        for (int i = 0; i < len; i++) {
            if (word[i] >= 'A' && word[i] <= 'Z') {
                word[i] += 32;
            } else if (word[i] >= 'a' && word[i] <= 'z') {
                word[i] -= 32;
            }
        }
    }
}

// 'q' Rule: Duplicate each character
__global__ void duplicateCharsKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (2 * len > MAX_LEN - 1) return;

        // Duplicate characters
        for (int i = len - 1; i >= 0; --i) {
            word[2 * i] = word[i];
            word[2 * i + 1] = word[i];
        }
        lengths[idx] = 2 * len;
        word[2 * len] = '\0';
    }
}

// 'r' Rule: Reverse Word
__global__ void reverseKernel(char *words, int *lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char *word = &words[idx * MAX_LEN];
        // Swap characters from beginning and end
        for (int i = 0; i < len / 2; i++) {
            char temp = word[i];
            word[i] = word[len - 1 - i];
            word[len - 1 - i] = temp;
        }
    }
}

// 'k' Rule: Swap first two characters
__global__ void swapFirstTwoKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len >= 2) {
            char temp = word[0];
            word[0] = word[1];
            word[1] = temp;
        }
    }
}

// 'K' Rule: Swap last two characters
__global__ void swapLastTwoKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len >= 2) {
            char temp = word[len-1];
            word[len-1] = word[len-2];
            word[len-2] = temp;
        }
    }
}

// 'd' Rule: Duplicate word
__global__ void duplicateWordKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len * 2 < MAX_LEN) {
            for (int i = 0; i < len; i++) {
                word[len + i] = word[i];
            }
            lengths[idx] = len * 2;
            word[len * 2] = '\0';
        }
    }
}

// 'f' Rule: Append reversed string
__global__ void reflectKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (2 * len > MAX_LEN - 1) return;

        // Append reversed characters
        for (int i = 0; i < len; ++i) {
            word[len + i] = word[len - 1 - i];
        }
        lengths[idx] = 2 * len;
        word[2 * len] = '\0';
    }
}

// '{' Rule: Rotate left (move first char to end)
__global__ void rotateLeftKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len <= 1) return;

        char first = word[0];
        for (int i = 0; i < len - 1; ++i) {
            word[i] = word[i + 1];
        }
        word[len - 1] = first;
    }
}

// '}' Rule: Rotate right (move last char to start)
__global__ void rotateRightKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len <= 1) return;

        char last = word[len - 1];
        for (int i = len - 1; i > 0; --i) {
            word[i] = word[i - 1];
        }
        word[0] = last;
    }
}

// '[' Rule: Delete first character
__global__ void deleteFirstKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len <= 0) return;
        for (int i = 0; i < len - 1; i++) {
            word[i] = word[i + 1];
        }
        lengths[idx] = len - 1;
        word[len - 1] = '\0';
    }
}

// ']' Rule: Delete last character
__global__ void deleteLastKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len <= 0) return;

        lengths[idx] = len - 1;
        word[len - 1] = '\0';
    }
}

// 'E' Rule: Title case (capitalize after spaces)
__global__ void titleCaseKernel(char* words, int* lengths, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];

        // Lowercase all
        for (int i = 0; i < len; ++i) {
            if (word[i] >= 'A' && word[i] <= 'Z') word[i] += 32;
        }

        // Capitalize first and after spaces
        if (len > 0 && word[0] >= 'a' && word[0] <= 'z') {
            word[0] -= 32;
        }
        for (int i = 1; i < len; ++i) {
            if (word[i - 1] == ' ' && word[i] >= 'a' && word[i] <= 'z') {
                word[i] -= 32;
            }
        }
    }
}

// 'T' Rule: Toggle case at position
__global__ void togglePositionKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos < 0 || pos >= len) return;

        if (word[pos] >= 'a' && word[pos] <= 'z') {
            word[pos] -= 32;
        } else if (word[pos] >= 'A' && word[pos] <= 'Z') {
            word[pos] += 32;
        }
    }
}

// 'p' Rule: Repeat word N times
__global__ void repeatWordKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        int newLen = len * (count + 1);
        if (newLen >= MAX_LEN) newLen = MAX_LEN - 1;

        for (int i = 1; i <= count && (i * len) < newLen; i++) {
            for (int j = 0; j < len && (i * len + j) < newLen; j++) {
                word[i * len + j] = word[j];
            }
        }
        lengths[idx] = newLen;
        word[newLen] = '\0';
    }
}

// 'D' Rule: Delete character at position
__global__ void deletePositionKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos < 0 || pos >= len) return;

        for (int i = pos; i < len - 1; ++i) {
            word[i] = word[i + 1];
        }
        lengths[idx] = len - 1;
        word[len - 1] = '\0';
    }
}

// 'z' Rule: Prepend first character multiple times
__global__ void prependFirstCharKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0 || count <= 0) return;

        int newLen = len + count;
        if (newLen > MAX_LEN - 1) newLen = MAX_LEN - 1;
        char first = word[0];

        // Shift existing characters
        for (int i = newLen - 1; i >= count; --i) {
            word[i] = word[i - count];
        }
        // Prepend first character
        for (int i = 0; i < count; ++i) {
            word[i] = first;
        }
        lengths[idx] = newLen;
        word[newLen] = '\0';
    }
}

// 'Z' Rule: Append last character N times
__global__ void appendLastCharKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0) return;

        char last = word[len - 1];
        int newLen = len + count;
        if (newLen >= MAX_LEN) newLen = MAX_LEN - 1;

        for (int i = len; i < newLen; i++) {
            word[i] = last;
        }
        lengths[idx] = newLen;
        word[newLen] = '\0';
    }
}

// ''' Rule: Truncate at position
__global__ void truncateAtKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        if (pos < len) {
            lengths[idx] = pos;
            words[idx * MAX_LEN + pos] = '\0';
        }
    }
}

// 's' Rule: replaces all occurrences of oldChar with newChar.
__global__ void substitutionKernel(char *words, int *lengths, char oldChar, char newChar, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char *word = &words[idx * MAX_LEN];
        for (int i = 0; i < len; i++) {
            if (word[i] == oldChar) {
                word[i] = newChar;
            }
        }
    }
}

// 'S' Rule: replaces first occurrence of oldChar with newChar.
__global__ void substitutionFirstKernel(char *words, int *lengths, char oldChar, char newChar, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char *word = &words[idx * MAX_LEN];
        for (int i = 0; i < len; i++) {
            if (word[i] == oldChar) {
                word[i] = newChar;
                break;
            }
        }
    }
}

// '$' Rule: Appends a given character to the end of each word.
__global__ void appendKernel(char *words, int *lengths, char appendChar, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        if (len < MAX_LEN - 1) {
            words[idx * MAX_LEN + len] = appendChar;
            words[idx * MAX_LEN + len + 1] = '\0';
            lengths[idx] = len + 1;
        }
    }
}

// '^' Rule: Prepends a given character to the start of each word.
__global__ void prependKernel(char *words, int *lengths, char prefixChar, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        if (len < MAX_LEN - 1) {
            char *word = &words[idx * MAX_LEN];
            // Shift the word one character to the right (including the terminator)
            for (int i = len; i >= 0; i--) {
                word[i + 1] = word[i];
            }
            word[0] = prefixChar;
            lengths[idx] = len + 1;
        }
    }
}

// 'y' Rule: Prepend substring
__global__ void prependSubstrKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        int prependLen = min(count, len);
        int newLen = len + prependLen;

        if (newLen >= MAX_LEN) {
            newLen = MAX_LEN - 1;
            prependLen = newLen - len;
        }

        // Shift right
        for (int i = newLen - 1; i >= prependLen; i--) {
            word[i] = word[i - prependLen];
        }
        // Copy prefix
        for (int i = 0; i < prependLen; i++) {
            word[i] = word[i + prependLen];
        }
        lengths[idx] = newLen;
        word[newLen] = '\0';
    }
}

// 'Y' Rule: Append substring
__global__ void appendSubstrKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        int appendLen = min(count, len);
        int newLen = len + appendLen;

        if (newLen >= MAX_LEN) {
            newLen = MAX_LEN - 1;
            appendLen = newLen - len;
        }

        for (int i = 0; i < appendLen; i++) {
            word[len + i] = word[len - appendLen + i];
        }
        lengths[idx] = newLen;
        word[newLen] = '\0';
    }
}

// 'L' Rule: Bitwise shift left at position
__global__ void bitShiftLeftKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos >= 0 && pos < len) {
            word[pos] = word[pos] << 1;
        }
    }
}

// 'R' Rule: Bitwise shift right at position
__global__ void bitShiftRightKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos >= 0 && pos < len) {
            word[pos] = word[pos] >> 1;
        }
    }
}

// '-' Rule: Decrement character at position
__global__ void decrementCharKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos >= 0 && pos < len) {
            word[pos]--;
        }
    }
}

// '+' Rule: Increment character at position
__global__ void incrementCharKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos >= 0 && pos < len) {
            word[pos]++;
        }
    }
}

// '@' Rule: Delete all instances of character
__global__ void deleteAllCharKernel(char* words, int* lengths, char target, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        int newIndex = 0;
        for (int i = 0; i < len; i++) {
            if (word[i] != target) {
                word[newIndex++] = word[i];
            }
        }
        lengths[idx] = newIndex;
        word[newIndex] = '\0';
    }
}

// '.' Rule: Swap with next character
__global__ void swapNextKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos >= 0 && pos < len - 1) {
            char temp = word[pos];
            word[pos] = word[pos + 1];
            word[pos + 1] = temp;
        }
    }
}

// ',' Rule: Swap with next character
__global__ void swapLastKernel(char* words, int* lengths, int pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (pos >= 0 && pos < len - 1) {
            char temp = word[pos];
            word[pos] = word[pos - 1];
            word[pos - 1] = temp;
        }
    }
}

// 'e' Rule: Title case with separator
__global__ void titleSeparatorKernel(char* words, int* lengths, char separator, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0) return;
        // Convert all characters to lowercase
        for (int i = 0; i < len; i++) {
            // Simple ASCII conversion: if character is A-Z, add 32.
            if (word[i] >= 'A' && word[i] <= 'Z') {
                word[i] = word[i] + 32;
            }
        }
        // Capitalize first character
        if (word[0] >= 'a' && word[0] <= 'z') {
            word[0] = word[0] - 32;
        }
        // Process trigger characters
        // -1 to prevent overflow in i+1 check and 1 to not double capitalize and gain small % performance
        // I can imagine this being undesired - feel free to report
        for (int i = 1; i < len - 1; ++i) {
            if (word[i] == separator) {
                word[i+1] = word[i+1] - 32;
            }
        }
    }
}

// 'i' Rule: Insert string at position
__global__ void insertKernel(char* words, int* lengths, int pos, char insert_char, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0 || pos > len) return;
        if (len + 1 >= MAX_LEN) return;

        // Shift characters right
        for (int i = len; i > pos; --i) {
            word[i] = word[i - 1];
        }
        word[pos] = insert_char;

        lengths[idx] = len + 1;
        word[len + 1] = '\0';
    }
}

// 'O' Rule: Omit Range M starting at pos N (ONM)
__global__ void OmitKernel(char* words, int* lengths, int pos, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len <= 0 || pos >= len) return;

        int delete_count = min(count, len - pos);
        for (int i = pos; i < len - delete_count; ++i) {
            word[i] = word[i + delete_count];
        }
        lengths[idx] = len - delete_count;
        word[len - delete_count] = '\0';
    }
}

// 'o' Rule: Overwrite at position
__global__ void overwriteKernel(char* words, int* lengths, int pos, char replace_char, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];

        // Check for empty string or invalid position
        if (len <= 0 || pos < 0 || pos >= len) return;

        // Overwrite single character
        word[pos] = replace_char;
    }
}

// '*' Rule: Swap any two
__global__ void swapAnyKernel(char* words, int* lengths, int pos, int replace_pos, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len == 0 || pos < 0 || replace_pos < 0 || pos >= len || replace_pos >= len) return;

        // Swap
        char temp = word[pos];
        word[pos] = word[replace_pos];
        word[replace_pos] = temp;
    }
}


// 'x' Rule: Delete all except a slice i.e. extract a substring
__global__ void extractKernel(char* words, int* lengths, int pos, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        if (len <= 0 || pos >= len) return;
        int keep_count = min(count, len - pos);
        // First, shift the kept slice to the beginning
        for (int i = 0; i < keep_count; ++i) {
            word[i] = word[pos + i];
        }
        // Truncate after the kept portion
        lengths[idx] = keep_count;
        word[keep_count] = '\0';
    }
}

// '<' Rule: Reject words longer than or equal to count
__global__ void rejectLessKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        if (lengths[idx] >= count) {
            lengths[idx] = 0;
            words[idx * MAX_LEN] = '\0';
        }
    }
}

// '>' Rule: Reject words shorter than or equal to count
__global__ void rejectGreaterKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        if (lengths[idx] <= count) {
            lengths[idx] = 0;
            words[idx * MAX_LEN] = '\0';
        }
    }
}

// '_' Rule: Reject words with exact length
__global__ void rejectEqualKernel(char* words, int* lengths, int count, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        if (lengths[idx] == count) {
            lengths[idx] = 0;
            words[idx * MAX_LEN] = '\0';
        }
    }
}

// '!' Rule: Reject words containing specific character
__global__ void rejectContainKernel(char* words, int* lengths, char contain_char, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];

        for (int i = 0; i < len; ++i) {
            if (word[i] == contain_char) {
                lengths[idx] = 0;
                word[0] = '\0';
                return;
            }
        }
    }
}

// '/' Rule: Reject words not containing specific character
__global__ void rejectNotContainKernel(char* words, int* lengths, char contain_char, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        bool found = false;

        for (int i = 0; i < len; ++i) {
            if (word[i] == contain_char) {
                found = true;
                break;
            }
        }

        if (!found) {
            lengths[idx] = 0;
            word[0] = '\0';
        }
    }
}

// '3' Rule: Toggle case after nth instance of character
__global__ void toggleWithNSeparatorKernel(char* words, int* lengths, char separator_char, int separator_num, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numWords) {
        int len = lengths[idx];
        char* word = &words[idx * MAX_LEN];
        int count = 0;
        int toggle_pos = -1;

        // Find nth instance
        for (int i = 0; i < len; ++i) {
            if (word[i] == separator_char) {
                if (count == separator_num) {
                    toggle_pos = i + 1;
                    break;
                }
                count++;
            }
        }

        // Toggle next character if found and within bounds
        if (toggle_pos > 0 && toggle_pos < len) {
            if (word[toggle_pos] >= 'a' && word[toggle_pos] <= 'z') {
                word[toggle_pos] = word[toggle_pos] - 32;
            } else if (word[toggle_pos] >= 'A' && word[toggle_pos] <= 'Z') {
                word[toggle_pos] = word[toggle_pos] + 32;
            }
        }
    }
}



//---------------------------------------------------------------------
// Host Wrappers to Launch Kernels
//---------------------------------------------------------------------
// l
DLL_EXPORT
void applyLowerCase(char *d_words, int *d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    lowerCaseKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// u
DLL_EXPORT
void applyUpperCase(char *d_words, int *d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    upperCaseKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// c
DLL_EXPORT
void applyCapitalize(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    capitalizeKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// C
DLL_EXPORT
void applyInvertCapitalize(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    invertCapitalizeKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// t
DLL_EXPORT
void applyToggleCase(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    toggleCaseKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// q
DLL_EXPORT
void applyDuplicateChars(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    duplicateCharsKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// r
DLL_EXPORT
void applyReverse(char *d_words, int *d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    reverseKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// k
DLL_EXPORT
void applySwapFirstTwo(char *d_words, int *d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    swapFirstTwoKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// K
DLL_EXPORT
void applySwapLastTwo(char *d_words, int *d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    swapLastTwoKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// d
DLL_EXPORT
void applyDuplicate(char *d_words, int *d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    duplicateWordKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// f
DLL_EXPORT
void applyReflect(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    reflectKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// {
DLL_EXPORT
void applyRotateLeft(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rotateLeftKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
 // }
DLL_EXPORT
void applyRotateRight(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rotateRightKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// [
DLL_EXPORT
void applyDeleteFirst(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    deleteFirstKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// ]
DLL_EXPORT
void applyDeleteLast(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    deleteLastKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// E
DLL_EXPORT
void applyTitleCase(char* d_words, int* d_lengths, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    titleCaseKernel<<<blocks, THREADS>>>(d_words, d_lengths, numWords);
    hipDeviceSynchronize();
}
// T
DLL_EXPORT
void applyTogglePosition(char* d_words, int* d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    togglePositionKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// p
DLL_EXPORT
void applyRepeatWord(char* d_words, int* d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    repeatWordKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}
// D
DLL_EXPORT
void applyDeletePosition(char* d_words, int* d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    deletePositionKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// z
DLL_EXPORT
void applyPrependFirstChar(char* d_words, int* d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    prependFirstCharKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}
// Z
DLL_EXPORT
void applyAppendLastChar(char* d_words, int* d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    prependFirstCharKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}
// '
DLL_EXPORT
void applyTruncateAt(char* d_words, int* d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    truncateAtKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// s
DLL_EXPORT
void applySubstitution(char *d_words, int *d_lengths, char oldChar, char newChar, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    substitutionKernel<<<blocks, THREADS>>>(d_words, d_lengths, oldChar, newChar, numWords);
    hipDeviceSynchronize();
}
// S
DLL_EXPORT
void applySubstitutionFirst(char *d_words, int *d_lengths, char oldChar, char newChar, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    substitutionFirstKernel<<<blocks, THREADS>>>(d_words, d_lengths, oldChar, newChar, numWords);
    hipDeviceSynchronize();
}
// $
DLL_EXPORT
void applyAppend(char *d_words, int *d_lengths, char appendChar, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    appendKernel<<<blocks, THREADS>>>(d_words, d_lengths, appendChar, numWords);
    hipDeviceSynchronize();
}
// ^
DLL_EXPORT
void applyPrepend(char *d_words, int *d_lengths, char prefixChar, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    prependKernel<<<blocks, THREADS>>>(d_words, d_lengths, prefixChar, numWords);
    hipDeviceSynchronize();
}
// y
DLL_EXPORT
void applyPrependPrefixSubstr(char *d_words, int *d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    prependSubstrKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}
// Y
DLL_EXPORT
void applyAppendSuffixSubstr(char *d_words, int *d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    appendSubstrKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}
// L
DLL_EXPORT
void applyBitShiftLeft(char *d_words, int *d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    bitShiftLeftKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// R
DLL_EXPORT
void applyBitShiftRight(char *d_words, int *d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    bitShiftRightKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// -
DLL_EXPORT
void applyDecrementChar(char *d_words, int *d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    decrementCharKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// +
DLL_EXPORT
void applyIncrementChar(char *d_words, int *d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    incrementCharKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// @
DLL_EXPORT
void applyDeleteAllChar(char *d_words, int *d_lengths, char deleteChar, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    deleteAllCharKernel<<<blocks, THREADS>>>(d_words, d_lengths, deleteChar, numWords);
    hipDeviceSynchronize();
}
// .
DLL_EXPORT
void applySwapNext(char *d_words, int *d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    swapNextKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// ,
DLL_EXPORT
void applySwapLast(char *d_words, int *d_lengths, int pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    swapLastKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, numWords);
    hipDeviceSynchronize();
}
// e
DLL_EXPORT
void applyTitleSeparator(char *d_words, int *d_lengths, char separator, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    titleSeparatorKernel<<<blocks, THREADS>>>(d_words, d_lengths, separator, numWords);
    hipDeviceSynchronize();
}
// i
DLL_EXPORT
void applyInsert(char* d_words, int* d_lengths, int pos, char insert_char, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    insertKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, insert_char, numWords);
    hipDeviceSynchronize();
}
// O
DLL_EXPORT
void applyOmit(char* d_words, int* d_lengths, int pos, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    OmitKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, count, numWords);
    hipDeviceSynchronize();
}
// o
DLL_EXPORT
void applyOverwrite(char* d_words, int* d_lengths, int pos, char replace_char, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    overwriteKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, replace_char, numWords);
    hipDeviceSynchronize();
}
// *
DLL_EXPORT
void applySwapAny(char* d_words, int* d_lengths, int pos, int replace_pos, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    swapAnyKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, replace_pos, numWords);
    hipDeviceSynchronize();
}
// x
DLL_EXPORT
void applyExtract(char* d_words, int* d_lengths, int pos, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    extractKernel<<<blocks, THREADS>>>(d_words, d_lengths, pos, count, numWords);
    hipDeviceSynchronize();
}
// <
DLL_EXPORT
void applyRejectLess(char* d_words, int* d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rejectLessKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}

// >
DLL_EXPORT
void applyRejectGreater(char* d_words, int* d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rejectGreaterKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}

// _
DLL_EXPORT
void applyRejectEqual(char* d_words, int* d_lengths, int count, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rejectEqualKernel<<<blocks, THREADS>>>(d_words, d_lengths, count, numWords);
    hipDeviceSynchronize();
}

// !
DLL_EXPORT
void applyRejectContain(char* d_words, int* d_lengths, char contain_char, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rejectContainKernel<<<blocks, THREADS>>>(d_words, d_lengths, contain_char, numWords);
    hipDeviceSynchronize();
}

// /
DLL_EXPORT
void applyRejectNotContain(char* d_words, int* d_lengths, char contain_char, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    rejectNotContainKernel<<<blocks, THREADS>>>(d_words, d_lengths, contain_char, numWords);
    hipDeviceSynchronize();
}

// 3
DLL_EXPORT
void applyToggleWithNSeparator(char* d_words, int* d_lengths, char separator_char, int separator_num, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    toggleWithNSeparatorKernel<<<blocks, THREADS>>>(d_words, d_lengths, separator_char, separator_num, numWords);
    hipDeviceSynchronize();
}

// Existing helper functions
DLL_EXPORT
void allocateOriginalDictMemoryOnGPU(
    char **d_originalDict, int **d_originalDictLengths,
    char *h_originalDict, int *h_originalDictLengths, int numWords
) {
    CUDA_CHECK(hipMalloc((void**)d_originalDict, numWords * MAX_LEN * sizeof(char)));
    CUDA_CHECK(hipMalloc((void**)d_originalDictLengths, numWords * sizeof(int)));

    CUDA_CHECK(hipMemcpy(*d_originalDict, h_originalDict, numWords * MAX_LEN * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(*d_originalDictLengths, h_originalDictLengths, numWords * sizeof(int), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

DLL_EXPORT
void allocateProcessedDictMemoryOnGPU(
    char **d_processedDict, int **d_processedDictLengths, int numWords
) {
    CUDA_CHECK(hipMalloc((void**)d_processedDict, numWords * MAX_LEN * sizeof(char)));
    CUDA_CHECK(hipMalloc((void**)d_processedDictLengths, numWords * sizeof(int)));
    hipDeviceSynchronize();
}

DLL_EXPORT
void allocateHashesMemoryOnGPU(
    uint64_t **d_originalHashes, int originalCount,
    uint64_t **d_compareHashes, int compareCount,
    uint64_t *h_originalHashes, uint64_t *h_compareHashes
) {
    CUDA_CHECK(hipMalloc((void**)d_originalHashes, originalCount * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc((void**)d_compareHashes, compareCount * sizeof(uint64_t)));

    CUDA_CHECK(hipMemcpy(*d_originalHashes, h_originalHashes, originalCount * sizeof(uint64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(*d_compareHashes, h_compareHashes, compareCount * sizeof(uint64_t), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

DLL_EXPORT
void ResetProcessedDictMemoryOnGPU(
    char **d_originalDict, int **d_originalDictLengths,
    char **d_processedDict, int **d_processedDictLengths,
    int numWords
) {
    CUDA_CHECK(hipMemcpy(*d_processedDict, *d_originalDict, numWords * MAX_LEN * sizeof(char), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(*d_processedDictLengths, *d_originalDictLengths, numWords * sizeof(int), hipMemcpyDeviceToDevice));
    hipDeviceSynchronize();
}

DLL_EXPORT
void copyMemoryBackToHost(
    char *h_processedDict, int *h_processedDictLengths,
    char **d_processedDict, int **d_processedDictLengths, int numWords) {
    CUDA_CHECK(hipMemcpy(h_processedDict, *d_processedDict, numWords * MAX_LEN * sizeof(char), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_processedDictLengths, *d_processedDictLengths, numWords * sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
}

DLL_EXPORT
void freeOriginalMemoryOnGPU(int *d_originalDict, int *d_originalDictLengths) {
    CUDA_CHECK(hipFree(d_originalDict));
    CUDA_CHECK(hipFree(d_originalDictLengths));
    hipDeviceSynchronize();
}

DLL_EXPORT
void freeProcessedMemoryOnGPU(char *d_processedDict, int *d_processedDictLengths, uint64_t *d_hitCount) {
    CUDA_CHECK(hipFree(d_processedDictLengths));
    CUDA_CHECK(hipFree(d_processedDict));
    CUDA_CHECK(hipFree(d_hitCount));
    hipDeviceSynchronize();
}

__global__ void xxhashKernel(char* d_processedDict, int* d_processedDictLengths , uint64_t seed, uint64_t* hashes, int numWords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWords) return;

    char* input = &d_processedDict[idx * MAX_LEN];
    hashes[idx] = xxhash64(input, d_processedDictLengths[idx], seed);
}

__global__ void xxhashWithCheckKernel(
    char* d_processedDict,
    int* d_processedLengths,
    uint64_t seed,
    const uint64_t* d_originalHashes,
    uint64_t originalCount,
    const uint64_t* d_compareHashes,
    uint64_t compareCount,
    uint64_t* hitCount
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= originalCount) return;

    // Compute hash
    char* word = &d_processedDict[idx * MAX_LEN];
    const uint64_t hash = xxhash64(word, d_processedLengths[idx], seed);
    // Atomic increment if valid hit
    const bool inCompare = binarySearchGPU(d_compareHashes, compareCount, hash);
    if (inCompare && !binarySearchGPU(d_originalHashes, originalCount, hash)) {
        atomicAdd(reinterpret_cast<unsigned long long*>(hitCount), 1ULL);
    }
}

// Exported function to launch the kernel
DLL_EXPORT
void computeXXHashes(char *d_processedDict, int *d_processedDictLengths, uint64_t seed, uint64_t* d_hashes, int numWords) {
    int blocks = (numWords + THREADS - 1) / THREADS;
    xxhashKernel<<<blocks, THREADS>>>(d_processedDict, d_processedDictLengths , seed, d_hashes, numWords);
    hipDeviceSynchronize();
}

DLL_EXPORT
void computeXXHashesWithCheck(
    char *d_processedDict, int *d_processedDictLengths, uint64_t seed,
    const uint64_t* d_originalHashes, int originalHashCount,
    const uint64_t* d_compareHashes, int compareHashCount,
    uint64_t* d_hitCount
) {
    int blocks = (originalHashCount + THREADS - 1) / THREADS;
    hipMemset(d_hitCount, 0, sizeof(uint64_t));
    hipDeviceSynchronize();
    xxhashWithCheckKernel<<<blocks, THREADS>>>(
        d_processedDict, d_processedDictLengths, seed,
        d_originalHashes, originalHashCount,
        d_compareHashes, compareHashCount,
        d_hitCount
    );
}

} // extern "C"